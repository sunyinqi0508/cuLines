#include "hip/hip_runtime.h"
#include "common.h"

#include <hip/hip_runtime.h>
#include ""

#include<stdio.h>
__device__ __host__
struct Struct {
	int a, b;
};

__global__ void test2(Struct *a) {
	a[threadIdx.x].a = blockIdx.x;
	a[threadIdx.x].b = threadIdx.x;
	printf("a\n");
}


void cudacall() {


	int *d_a, *a = new int[64];
	Struct *s = new Struct[4], *d_s;
	hipMalloc(&d_s, 4 * sizeof(Struct));
	test2 << <4, 4 >> > (d_s); 
	hipMemcpy(s, d_s, 4 * sizeof(Struct), hipMemcpyDeviceToHost);
	for(int i = 0;i<4;i++)
		printf("%d %d \n", s[i].a, s[i].b);

}