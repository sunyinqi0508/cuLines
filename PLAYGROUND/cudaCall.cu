#include "hip/hip_runtime.h"
#include "common.h"

#include <hip/hip_runtime.h>
#include ""

#include<stdio.h>
__global__ void test2(int *a=0) {
	a[threadIdx.x] = blockIdx.x;
	printf("a\n");
}


void cudacall() {


	int *d_a, *a = new int[64];
	hipMalloc(&d_a, 64);
	test2 << <4, 4 >> > (d_a); 
	hipMemcpy(a, d_a, 16, hipMemcpyDeviceToHost);
	printf("%d", a[3]);

}