#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdint.h>
#include "cuStubs.cuh"
#include "Vector.h"


template<typename T>
__device__
inline T constexpr pow2(const T v) noexcept { return v * v; }

__global__
void cuLSH( //lv.1 search

	const GPU_Lsh_Func *funcs, const GPU_HashTable *hashtable, const Table_Contents *table_contents, 
	const GPU_Segments *segments, int *__restrict__ results,  float* temp, 
	const float *f_stramlines, const int* lineoffsets, const int n_lines, const int n_pts

) {
	temp += (blockIdx.x * blockDim.x + threadIdx.x)*n_lines;
	
	bool sgn = false;
	for (int i = blockIdx.x; i < n_lines; i += gridDim.x) {
		const float* streamline = f_stramlines + lineoffsets[i];
		for (int j = threadIdx.x; j < lineoffsets[i + 1] - lineoffsets[i]; j += blockDim.x) {
			int cnt_result = 0;
			int ptnum = lineoffsets[i] + j;
			for (int t = 0; t < TABLESIZE; t++)
			{
				int64_t fingerprint1 = 0, fingerprint2 = 0;
				for (int f = 0; f < K; f++) {

					const GPU_Lsh_Func curr_func = funcs[hashtable[t].LSHFuncs[f]];
					const int n_buckets = curr_func.n_buckets;
					const int func_val = curr_func(streamline + j*3);
					int64_t tmp_fp1 = hashtable[t].r1[f] * func_val;
					int64_t tmp_fp2 = hashtable[t].r1[f] * func_val;
					tmp_fp1 = 5 * (tmp_fp1 >> 32) + (tmp_fp1 & 0xffffffff);
					tmp_fp2 = 5 * (tmp_fp2 >> 32) + (tmp_fp2 & 0xffffffff);

					fingerprint1 += (tmp_fp1 >> 32) ? (tmp_fp1 - Prime) : tmp_fp1;
					fingerprint2 += (tmp_fp2 >> 32) ? (tmp_fp2 - Prime) : tmp_fp2;

					fingerprint1 = (fingerprint1 >> 32) ? (fingerprint1 - Prime) : fingerprint1;
					fingerprint2 = (fingerprint2 >> 32) ? (fingerprint2 - Prime) : fingerprint2;
				}
				fingerprint1 %= TABLESIZE;
				fingerprint2 %= Prime;
				
				int k = hashtable[t].table_offsets[fingerprint1];
				for (; k < hashtable[t].table_offsets[fingerprint1 + 1]; k++) {
					if (table_contents[k].fingerprint2 == fingerprint2) //optimize: group tables accroding to fingerprint2;
					{
						const int segment = table_contents[k].segment;
						const int line = segments[segment].line;
						if(temp[line] > 0 && sgn)
							results[ptnum*maxNN + cnt_result++] = segment;
						else {
							const float dist = temp[line];
							float this_dist = 0;
#pragma unroll
							for (int _dim = 0; _dim < 3; _dim++)
								this_dist += pow2(segments[segment].centroid[_dim] - streamline[j * 3 + _dim]);
							if (this_dist > dist) {
								const int res_pos = *reinterpret_cast<int*>(temp + line) & 0x1f;
								int *i_dist = reinterpret_cast<int*> (&this_dist);
								*i_dist = *i_dist & 0xffffffe0 + cnt_result;
								temp[line] = sgn ? -this_dist : this_dist;
								results[ptnum*maxNN + res_pos] = segment;
							}
						}
					}
					if (cnt_result >= maxNN) {
						goto finalize;
					}
				}
			}
		finalize:
			if (cnt_result < maxNN)
				results[ptnum*maxNN + cnt_result] = -1;
			sgn = !sgn;
		}

	}
}


__global__
void cuLineHashing(//Lv.2 search

	int *__restrict__ results,
	const GPU_SegmentsLv2 *segs_lv2, const float *f_stremlines, const int* lineoffsets,
	const short* lv2_buckets, const int n_lines, const int n_pts

) {

	int i = threadIdx.x;
	for (; i < n_lines; i+= blockDim.x) {
		int j = blockIdx.x;
		for (; j < lineoffsets[i + 1]; j += gridDim.x) {
			
			int ptnum = lineoffsets[i] + j;

			for (int k = 0; k < maxNN; k++) 
				if (results[ptnum*maxNN + k] > 0)
				{
					const int this_seg = results[ptnum * maxNN + k];
					const int ptoffset = segs_lv2[this_seg].bucket_pos_offset;
					const int bucket_begin = segs_lv2[this_seg].bucket_pos_offset;
					float projection = 0;
#pragma unroll 
					for (int _dim = 0; _dim < 3; _dim++) 
						projection += 
							(f_stremlines[ptnum * 3 + _dim] - segs_lv2[this_seg].origin[_dim]) * segs_lv2[this_seg].projector[_dim];
					
					int bucket = std::floor(projection);
					if (projection < 0)
						bucket = 0;
					else if (projection > segs_lv2[this_seg].width - 1)
						bucket = segs_lv2[this_seg].width - 1;

					results[ptnum * maxNN + k] = segs_lv2[this_seg].line << 16 + (ptoffset + lv2_buckets[bucket_begin + bucket]);//n_lines < 65535 && pt_on_line < 65535
				}
				else
					break;


		}
	}
}

__global__ 
void cuHeapify() {

}


__global__
void cuSimilarity(

	_out_ float *variation, _out_ float* distances, _out_ int*points,
	_in_ const float *f_streamlines, _in_ const int * lineoffsets, _in_ const int* results,
	const int n_lines, const int n_pts

) {

	int i = threadIdx.x;
	for (; i < n_lines; i += blockDim.x) {
		int j = blockIdx.x;
		for (; j < lineoffsets[i + 1]; j += gridDim.x) {
			const int ptnum = lineoffsets[i] + j;
			for (int k = 0; k < maxNN; k++)
				if (results[ptnum*maxNN + k] != -1) {

					const unsigned int targetline = ((unsigned)results[ptnum*maxNN + k]) >> 16;
					const unsigned int targetpt_on_line = ((unsigned)results[ptnum*maxNN + k] & 0xffff);
					const unsigned int target_ptnum = lineoffsets[targetline] + targetpt_on_line;
					
					int begin = lineoffsets[i] + (j > similarity_window) ? (j - similarity_window) : 0;
					int end = lineoffsets[i] + j + similarity_window;
					end = (end >= lineoffsets[i + 1]) ? lineoffsets[i + 1] - 1 : end;

					int forward = ptnum - begin;
					int backward = end - ptnum;
					   
					forward = __macro_min(targetpt_on_line, forward);
					backward = __macro_min(lineoffsets[targetline + 1] - lineoffsets[targetline] - targetpt_on_line - 1, backward);

					float center_dist = 0;

#pragma unroll 
					for (int _dim = 0; _dim < 3; _dim++)
						center_dist += pow2(f_streamlines[ptnum*3 + _dim] - f_streamlines[target_ptnum*3 + _dim]);
					center_dist = sqrtf(center_dist);


					float _variation = 0;
					int start_this = ptnum - forward, start_target = target_ptnum - forward;
					for (; start_this < ptnum; start_this++, start_target++) {
						float _dist = 0;
#pragma unroll 
						for (int _dim = 0; _dim < 3; _dim++) 
							_dist += pow2(f_streamlines[start_this * 3 + _dim] - f_streamlines[start_target * 3 + _dim]);
						_variation += pow2(center_dist - sqrtf(_dist));
					}

					for (; start_this < ptnum + backward; start_this++, start_target++) {
						float _dist = 0;
#pragma unroll 
						for (int _dim = 0; _dim < 3; _dim++)
							_dist += pow2(f_streamlines[start_this * 3 + _dim] - f_streamlines[start_target * 3 + _dim]);
						_variation += pow2(center_dist - sqrtf(_dist));
					}

					const int interval = backward + forward - 1;
					if (interval > 0)
						_variation /= interval;
					else
						_variation = 0;

					distances[ptnum*maxNN + k] = center_dist;
					variation[ptnum * maxNN + k] = _variation;
				}
				else break;

		}
	}

}

void cudaLauncher();

namespace cudadevice_variables {
	extern GPU_SegmentsLv2* segslv2;  //Lv.2 hash projector 
	extern GPU_Segments* segs;//centroid + line No. for Lv.1 LSH
	extern float* l2buckets;
	extern GPU_HashTable *d_hash;
	extern GPU_Lsh_Func *d_funcs;
}
